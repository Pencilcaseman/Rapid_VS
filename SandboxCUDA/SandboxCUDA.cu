// #define RAPID_NO_BLAS
// #define RAPID_NO_AMP
// #define RAPID_NO_GRAPHICS
// #define RAPID_CHECK_NAN

#define RAPID_CUDA

#include <iostream>
#include <rapid/rapid.h>

int main()
{
	using rapid::ndarray::CPU;
	using rapid::ndarray::GPU;

	// auto arr = rapid::ndarray::Array<float, GPU>::fromData({{1, 2}, {3, 4}});
	// 
	// std::cout << arr << "\n\n";
	// std::cout << arr + arr << "\n\n";
	// std::cout << arr - arr << "\n\n";
	// std::cout << arr * arr << "\n\n";
	// std::cout << arr / arr << "\n\n";
	// 
	// std::cout << "\n\n";
	// 
	// std::cout << arr + 10 << "\n\n";
	// std::cout << arr - 10 << "\n\n";
	// std::cout << arr * 10 << "\n\n";
	// std::cout << arr / 10 << "\n\n";
	// 
	// std::cout << "\n\n";
	// 
	// std::cout << arr << "\n\n";
	// arr += rapid::ndarray::Array<float, GPU>::fromData({{1, 2}, {3, 4}});
	// std::cout << arr << "\n\n";
	// arr -= rapid::ndarray::Array<float, GPU>::fromData({{1, 2}, {3, 4}});
	// std::cout << arr << "\n\n";
	// arr *= rapid::ndarray::Array<float, GPU>::fromData({{1, 2}, {3, 4}});
	// std::cout << arr << "\n\n";
	// arr /= rapid::ndarray::Array<float, GPU>::fromData({{1, 2}, {3, 4}});
	// std::cout << arr << "\n\n";
	// 
	// std::cout << "\n\n";
	// 
	// std::cout << arr << "\n\n";
	// arr += 10;
	// std::cout << arr << "\n\n";
	// arr -= 10;
	// std::cout << arr << "\n\n";
	// arr *= 10;
	// std::cout << arr << "\n\n";
	// arr /= 10;
	// std::cout << arr << "\n\n";
	// 
	// std::cout << "\n\n\n\n";

	auto lhs = rapid::ndarray::Array<float, GPU>::fromData({{1, 2, 3}, {4, 5, 6}, {7, 8, 9}});
	auto rhs = rapid::ndarray::Array<float, GPU>::fromData({{1, 2}, {3, 4}, {5, 6}});

	std::cout << lhs << "\n\n";
	std::cout << rhs << "\n\n";
	std::cout << lhs.dot(rhs) << "\n\n";

	rhs[1][1] = 12345;
	rhs[0] = rapid::ndarray::Array<float, GPU>::fromData({123, 456});

	std::cout << rhs << "\n\n";
	std::cout << "Test: " << rhs[1][0] << "\n";

	/*
	{
		std::cout << "Timing GPU<float>\n";
		auto speedTestGPU = rapid::ndarray::Array<float, GPU>({1000, 1000});

		START_TIMER(0, 10000);
		auto res = speedTestGPU.dot(speedTestGPU);
		END_TIMER(0);

		std::cout << "Timing CPU<float>\n";
		auto speedTestCPU = rapid::ndarray::Array<float, CPU>({1000, 1000});

		START_TIMER(1, 100);
		auto res = speedTestCPU.dot(speedTestCPU);
		END_TIMER(1);
	}

	{
		std::cout << "Timing GPU<double>\n";
		auto speedTestGPU = rapid::ndarray::Array<double, GPU>({1000, 1000});

		START_TIMER(0, 1000);
		auto res = speedTestGPU.dot(speedTestGPU);
		END_TIMER(0);

		std::cout << "Timing CPU<double>\n";
		auto speedTestCPU = rapid::ndarray::Array<double, CPU>({1000, 1000});

		START_TIMER(1, 100);
		auto res = speedTestCPU.dot(speedTestCPU);
		END_TIMER(1);
	}
	*/

	std::cout << "\n\n\n\n\n";
	auto a = rapid::ndarray::Array<float, GPU>({2, 2, 2});
	a.fill(0);
	auto b = rapid::ndarray::Array<float, GPU>::fromData({{1, 2}, {3, 4}});

	std::cout << a << "\n\n";
	std::cout << b << "\n\n";

	a[0] = b;
	std::cout << a << "\n\n";

	return 0;
}
